#include "hip/hip_runtime.h"
/**
 * SHA256d
 * by tpruvot@github - 2017
 */

#include <miner.h>
#include <cuda_helper.h>
#include <openssl/sha.h>


static bool init[MAX_GPUS] = { 0 };
extern void sha256d_init(int thr_id);
extern void sha256d_free(int thr_id);
extern void sha256d_hash_80(int thr_id, uint32_t threads, uint32_t startNonce, const uint32_t* const ms, uint32_t merkle, uint32_t time, uint32_t compacttarget, uint32_t* resNonces);

extern void sha256d_midstate(const uint32_t* data, uint32_t* midstate);

uint32_t rrot(uint32_t x, unsigned int n)
{
	return (x >> n) | (x << (32 - n));
}

// CPU Check
extern "C" void sha256d_hash(void* output, const void* input)
{
	unsigned char _ALIGN(64) hash[64];
	SHA256_CTX sha256;

	SHA256_Init(&sha256);
	SHA256_Update(&sha256, (unsigned char*)input, 80);
	SHA256_Final(hash, &sha256);

	SHA256_Init(&sha256);
	SHA256_Update(&sha256, hash, 32);
	SHA256_Final((unsigned char*)output, &sha256);
}

void sha256d_opt_hash(uint32_t* output, const uint32_t* data, uint32_t nonce, const uint32_t* midstate)
{
	int i;
	uint32_t s0, s1, t1, t2, maj, ch, a, b, c, d, e, f, g, h;
	uint32_t w[64];

	const uint32_t k[64] = {
		0x428a2f98, 0x71374491, 0xb5c0fbcf, 0xe9b5dba5, 0x3956c25b, 0x59f111f1, 0x923f82a4, 0xab1c5ed5,
		0xd807aa98, 0x12835b01, 0x243185be, 0x550c7dc3, 0x72be5d74, 0x80deb1fe, 0x9bdc06a7, 0xc19bf174,
		0xe49b69c1, 0xefbe4786, 0x0fc19dc6, 0x240ca1cc, 0x2de92c6f, 0x4a7484aa, 0x5cb0a9dc, 0x76f988da,
		0x983e5152, 0xa831c66d, 0xb00327c8, 0xbf597fc7, 0xc6e00bf3, 0xd5a79147, 0x06ca6351, 0x14292967,
		0x27b70a85, 0x2e1b2138, 0x4d2c6dfc, 0x53380d13, 0x650a7354, 0x766a0abb, 0x81c2c92e, 0x92722c85,
		0xa2bfe8a1, 0xa81a664b, 0xc24b8b70, 0xc76c51a3, 0xd192e819, 0xd6990624, 0xf40e3585, 0x106aa070,
		0x19a4c116, 0x1e376c08, 0x2748774c, 0x34b0bcb5, 0x391c0cb3, 0x4ed8aa4a, 0x5b9cca4f, 0x682e6ff3,
		0x748f82ee, 0x78a5636f, 0x84c87814, 0x8cc70208, 0x90befffa, 0xa4506ceb, 0xbef9a3f7, 0xc67178f2
	};
	const uint32_t hc[8] = {
		0x6a09e667U, 0xbb67ae85U, 0x3c6ef372U, 0xa54ff53aU,
		0x510e527fU, 0x9b05688cU, 0x1f83d9abU, 0x5be0cd19U
	};

	for (i = 0; i <= 15; ++i)
	{
		w[i] = data[i + 16];
	}
	w[3] = nonce;
	for (i = 16; i <= 63; ++i)
	{
		s0 = rrot(w[i - 15], 7) ^ rrot(w[i - 15], 18) ^ (w[i - 15] >> 3);
		s1 = rrot(w[i - 2], 17) ^ rrot(w[i - 2], 19) ^ (w[i - 2] >> 10);
		w[i] = w[i - 16] + s0 + w[i - 7] + s1;
	}
	a = midstate[0];
	b = midstate[1];
	c = midstate[2];
	d = midstate[3];
	e = midstate[4];
	f = midstate[5];
	g = midstate[6];
	h = midstate[7];
	for (i = 0; i <= 63; ++i)
	{
		s0 = rrot(a, 2) ^ rrot(a, 13) ^ rrot(a, 22);
		maj = (a & b) ^ (a & c) ^ (b & c);
		t2 = s0 + maj;
		s1 = rrot(e, 6) ^ rrot(e, 11) ^ rrot(e, 25);
		ch = (e & f) ^ ((~e) & g);
		t1 = h + s1 + ch + k[i] + w[i];
		h = g;
		g = f;
		f = e;
		e = d + t1;
		d = c;
		c = b;
		b = a;
		a = t1 + t2;
	}
	w[0] = a + midstate[0];
	w[1] = b + midstate[1];
	w[2] = c + midstate[2];
	w[3] = d + midstate[3];
	w[4] = e + midstate[4];
	w[5] = f + midstate[5];
	w[6] = g + midstate[6];
	w[7] = h + midstate[7];
	w[8] = 0x80000000U;
	for (i = 9; i <= 14; ++i)
		w[i] = 0U;
	w[15] = 0x100U;
	for (i = 16; i <= 63; ++i)
	{
		s0 = rrot(w[i - 15], 7) ^ rrot(w[i - 15], 18) ^ (w[i - 15] >> 3);
		s1 = rrot(w[i - 2], 17) ^ rrot(w[i - 2], 19) ^ (w[i - 2] >> 10);
		w[i] = w[i - 16] + s0 + w[i - 7] + s1;
	}
	a = hc[0];
	b = hc[1];
	c = hc[2];
	d = hc[3];
	e = hc[4];
	f = hc[5];
	g = hc[6];
	h = hc[7];
	for (i = 0; i <= 63; ++i)
	{
		s0 = rrot(a, 2) ^ rrot(a, 13) ^ rrot(a, 22);
		maj = (a & b) ^ (a & c) ^ (b & c);
		t2 = s0 + maj;
		s1 = rrot(e, 6) ^ rrot(e, 11) ^ rrot(e, 25);
		ch = (e & f) ^ ((~e) & g);
		t1 = h + s1 + ch + k[i] + w[i];
		h = g;
		g = f;
		f = e;
		e = d + t1;
		d = c;
		c = b;
		b = a;
		a = t1 + t2;
	}
	be32enc(&output[0], a + hc[0]);
	be32enc(&output[1], b + hc[1]);
	be32enc(&output[2], c + hc[2]);
	be32enc(&output[3], d + hc[3]);
	be32enc(&output[4], e + hc[4]);
	be32enc(&output[5], f + hc[5]);
	be32enc(&output[6], g + hc[6]);
	be32enc(&output[7], h + hc[7]);
}

extern "C" int scanhash_sha256d(int thr_id, struct work* work, uint32_t max_nonce, unsigned long *hashes_done)
{
	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;
	const uint32_t first_nonce = pdata[19];
	uint32_t throughput = cuda_default_throughput(thr_id, 1U << 25);
	if (init[thr_id]) throughput = min(throughput, (max_nonce - first_nonce));

	if (opt_benchmark)
		((uint32_t*)ptarget)[7] = 0x03;

	if (!init[thr_id])
	{
		hipSetDevice(device_map[thr_id]);
		if (opt_cudaschedule == -1 && gpu_threads == 1) {
			hipDeviceReset();
			// reduce cpu usage
			hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
			CUDA_LOG_ERROR();
		}
		gpulog(LOG_INFO, thr_id, "Intensity set to %g, %u cuda threads", throughput2intensity(throughput), throughput);

		sha256d_init(thr_id);

		init[thr_id] = true;
	}

	uint32_t ms[8];
	sha256d_midstate(pdata, ms);

	do {
		// Hash with CUDA
		*hashes_done = pdata[19] - first_nonce + throughput;

		sha256d_hash_80(thr_id, throughput, pdata[19], ms, pdata[16], pdata[17], pdata[18], work->nonces);
		if (work->nonces[0] != UINT32_MAX)
		{
			uint32_t _ALIGN(64) vhash[8];

			sha256d_opt_hash(vhash, pdata, work->nonces[0], ms);
			if (vhash[7] <= ptarget[7] && fulltest(vhash, ptarget)) {
				work->valid_nonces = 1;
				work_set_target_ratio(work, vhash);
				if (work->nonces[1] != UINT32_MAX) {
					sha256d_opt_hash(vhash, pdata, work->nonces[1], ms);
					if (vhash[7] <= ptarget[7] && fulltest(vhash, ptarget)) {
						work->valid_nonces++;
						bn_set_target_ratio(work, vhash, 1);
					}
					pdata[19] = max(work->nonces[0], work->nonces[1]) + 1;
				} else {
					pdata[19] = work->nonces[0] + 1;
				}
				return work->valid_nonces;
			}
			else if (vhash[7] > ptarget[7]) {
				gpu_increment_reject(thr_id);
				if (!opt_quiet) {
					gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU!", work->nonces[0]);
				}
				pdata[19] = work->nonces[0] + 1;
				continue;
			}
		}

		if ((uint64_t) throughput + pdata[19] >= max_nonce) {
			pdata[19] = max_nonce;
			break;
		}

		pdata[19] += throughput;

	} while (!work_restart[thr_id].restart);

	*hashes_done = pdata[19] - first_nonce;

	return 0;
}

// cleanup
extern "C" void free_sha256d(int thr_id)
{
	if (!init[thr_id])
		return;

	hipDeviceSynchronize();

	sha256d_free(thr_id);

	init[thr_id] = false;

	hipDeviceSynchronize();
}
